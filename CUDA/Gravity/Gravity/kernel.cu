#include "hip/hip_runtime.h"
﻿/*
CUDA code for calculating using only CUDA thread
*/
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<math.h>

#define N 1000
#define G 10
#define DT .1

__global__ void kernelGravity(double*, double*, double*, double*);

int main()
{
	hipSetDevice(0);

	size_t size;

	double* d_m;
	size = sizeof(double) * N;
	hipMalloc(&d_m, size);

	double* d_a;
	size = sizeof(double) * N * N * 3;
	hipMalloc(&d_a, size);

	double* d_v;
	size = sizeof(double) * N * 3;
	hipMalloc(&d_v, size);

	double* d_pos;
	hipMalloc(&d_pos, size);

	//TODO	hipMemcpy

	kernelGravity << <1, N >> > (d_m, d_a, d_v, d_pos);

	hipDeviceReset();

	return 0;
}

__global__ void kernelGravity(double* m, double* a, double* v, double* pos) {
	int i = threadIdx.x;
	double r_sqr;
	for (int j = 0; j < N; j++) {
		r_sqr = (pos[i] - pos[j]) * (pos[i] - pos[j])
			+ (pos[i + N] - pos[j + N]) * (pos[i + N] - pos[j + N])
			+ (pos[i + N * 2] - pos[j + N * 2]) * (pos[i + N * 2] - pos[j + N * 2]);

		a[i * N + j] = G * (m[j]) / (r_sqr)
			* (pos[i] - pos[j]) / (sqrt(r_sqr));
		a[i * N + j + N * N] = G * (m[j]) / (r_sqr)
			* (pos[i + N] - pos[j + N]) / (sqrt(r_sqr));
		a[i * N + j + N * N * 2] = G * (m[j]) / (r_sqr)
			* (pos[i + N * 2] - pos[j + N * 2]) / (sqrt(r_sqr));
	}

	for (int j = 0; j < N; j++) {
		if (i != j) {
			v[i] += DT * a[i * N + j];
			v[i + N] += DT * a[i * N + j + N * N];
			v[i + N * 2] += DT * a[i * N + j + N * N * 2];
		}
	}

	__syncthreads();

	pos[i] += DT * v[i];
	pos[i + N] += DT * v[i + N];
	pos[i + N * 2] += DT * v[i + N * 2];

}