#include "hip/hip_runtime.h"
﻿/*
CUDA code for calculating using only CUDA thread
*/
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "../data_util/data_util.h"

#define N 1000
#define G 1
#define DT 1

__global__ void kernelGravity(double*, double*, double*, double*, double*);

int main()
{
	size_t size;

	// host variable
	size = sizeof(double) * N;
	double* m = (double*)malloc(size);
	size = sizeof(double) * N * N * 3;
	double* a = (double*)malloc(size);
	size = sizeof(double) * N * 3;
	double* v = (double*)malloc(size);
	double* pos = (double*)malloc(size);

	// check variable
	if (
		m == NULL ||
		a == NULL ||
		v == NULL ||
		pos == NULL
		) {
		fprintf(stderr, "Err: Malloc Failed");
		return -1;
	}

	// set variable
	if (readDouble("../data_util/test/n1000/m.double", m) != N) {
		fprintf(stderr, "Err: Can not read m.double");
		return -1;
	}

	size = sizeof(double) * N * N * 3;
	memset(a, 0, size);

	size = sizeof(double) * N * 3;
	memset(v, 0, size);

	if (readDouble("../data_util/test/n1000/x.double", pos) != N) {
		fprintf(stderr, "Err: Can not read x.double");
		return -1;
	}
	if (readDouble("../data_util/test/n1000/y.double", pos + N) != N) {
		fprintf(stderr, "Err: Can not read y.double");
		return -1;
	}
	if (readDouble("../data_util/test/n1000/z.double", pos + N * 2) != N) {
		fprintf(stderr, "Err: Can not read z.double");
		return -1;
	}

	// device(CUDA) variable
	hipSetDevice(0);

	double* d_m;
	size = sizeof(double) * N;
	hipMalloc(&d_m, size);

	double* d_a;
	size = sizeof(double) * N * N * 3;
	hipMalloc(&d_a, size);

	double* d_v;
	size = sizeof(double) * N * 3;
	hipMalloc(&d_v, size);

	double* d_pos;
	hipMalloc(&d_pos, size);

	double* d_result;
	hipMalloc(&d_result, size);

	// performance metric
	hipEvent_t start, memcpy, end;
	hipEventCreate(&start);
	hipEventCreate(&memcpy);
	hipEventCreate(&end);
	hipEventRecord(start);	// record event

	// hipMemcpy
	size = sizeof(double) * N;
	hipMemcpy(d_m, m, size, hipMemcpyHostToDevice);
	size = sizeof(double) * N * N * 3;
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	size = sizeof(double) * N * 3;
	hipMemcpy(d_v, v, size, hipMemcpyHostToDevice);
	hipMemcpy(d_pos, pos, size, hipMemcpyHostToDevice);
	hipEventRecord(memcpy);	// record event

	// launch kernel
	for (int i = 0; i < 10; i++) {
		kernelGravity << <1, N >> > (d_m, d_a, d_v, d_pos, d_result);
		//hipDeviceSynchronize();
		hipMemcpy(d_pos, d_result, size, hipMemcpyDeviceToDevice);
		//hipDeviceSynchronize();
	}

	// hipMemcpy result from device
	hipMemcpy(pos, d_pos, size, hipMemcpyDeviceToHost);
	hipEventRecord(end);	// record event
	hipDeviceSynchronize();

	// print performace metrics
	float milisecondsStoM = 0;
	hipEventElapsedTime(&milisecondsStoM, start, memcpy);
	fprintf(stdout, "Time from start to memcpy:\n\t%f\n", milisecondsStoM);
	float milisecondsStoE = 0;
	hipEventElapsedTime(&milisecondsStoE, start, end);
	fprintf(stdout, "Time from start to end:\n\t%f\n", milisecondsStoE);

	// write results
	char writeFileX[] = "./cuda_x.double";
	writeDouble(writeFileX, pos, N);
	char writeFileY[] = "./cuda_y.double";
	writeDouble(writeFileY, pos + N, N);
	char writeFileZ[] = "./cuda_z.double";
	writeDouble(writeFileZ, pos + N * 2, N);

	// free memory
	hipFree(d_m);
	hipFree(d_a);
	hipFree(d_v);
	hipFree(d_pos);
	hipFree(d_result);

	free(m);
	free(a);
	free(v);
	free(pos);

	hipDeviceReset();

	return 0;
}

__global__ void kernelGravity(double* m, double* a, double* v, double* pos, double* result) {
	int i = threadIdx.x;
	double r_sqr;
	double r;
	for (int j = 0; j < N; j++) {
		r_sqr
			= (pos[i] - pos[j]) * (pos[i] - pos[j])
			+ (pos[i + N] - pos[j + N]) * (pos[i + N] - pos[j + N])
			+ (pos[i + N * 2] - pos[j + N * 2]) * (pos[i + N * 2] - pos[j + N * 2]);
		r = sqrt(r_sqr);
		a[i * N + j]
			= G * (m[j]) / (r_sqr) * (pos[j] - pos[i]) / r;
		a[i * N + j + N * N]
			= G * (m[j]) / (r_sqr) * (pos[j + N] - pos[i + N]) / r;
		a[i * N + j + N * N * 2]
			= G * (m[j]) / (r_sqr) * (pos[j + N * 2] - pos[i + N * 2]) / r;
	}

	for (int j = 0; j < N; j++) {
		if (i != j) {
			v[i] += DT * a[i * N + j];
			v[i + N] += DT * a[i * N + j + N * N];
			v[i + N * 2] += DT * a[i * N + j + N * N * 2];
		}
	}

	result[i] = pos[i] + DT * v[i];
	result[i + N] = pos[i + N] + DT * v[i + N];
	result[i + N * 2] = pos[i + N * 2] + DT * v[i + N * 2];

}