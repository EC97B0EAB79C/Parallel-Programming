#include "hip/hip_runtime.h"
﻿/*
CUDA code for calculating using only CUDA thread
*/
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N 1000
#define G 10
#define DT .1

__global__ void kernelGravity(double*, double*, double*, double*);

int main()
{
	size_t size;

	// host variable
	size = sizeof(double) * N;
	double* m = (double*)malloc(size);
	size = sizeof(double) * N * N * 3;
	double* a = (double*)malloc(size);
	size = sizeof(double) * N * 3;
	double* v = (double*)malloc(size);
	double* pos = (double*)malloc(size);

	//TODO set variable

	// device(CUDA) variable
	hipSetDevice(0);

	double* d_m;
	size = sizeof(double) * N;
	hipMalloc(&d_m, size);

	double* d_a;
	size = sizeof(double) * N * N * 3;
	hipMalloc(&d_a, size);

	double* d_v;
	size = sizeof(double) * N * 3;
	hipMalloc(&d_v, size);

	double* d_pos;
	hipMalloc(&d_pos, size);

	// hipMemcpy
	size = sizeof(double) * N;
	hipMemcpy(d_m, m, size, hipMemcpyHostToDevice);
	size = sizeof(double) * N * N * 3;
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	size = sizeof(double) * N * 3;
	hipMemcpy(d_v, v, size, hipMemcpyHostToDevice);
	hipMemcpy(d_pos, pos, size, hipMemcpyHostToDevice);

	// block, thread size
	int blockDim = 256;
	int gridDim = (N + blockDim + 1) / blockDim;


	// launch kernel
	kernelGravity << <gridDim, blockDim >> > (d_m, d_a, d_v, d_pos);

	// sync device
	hipDeviceSynchronize();

	// hipMemcpy result from device
	hipMemcpy(pos, d_pos, size, hipMemcpyDeviceToHost);

	// free memory
	hipFree(d_m);
	hipFree(d_a);
	hipFree(d_v);
	hipFree(d_pos);

	free(m);
	free(a);
	free(v);
	free(pos);

	hipDeviceReset();

	return 0;
}

__global__ void kernelGravity(double* m, double* a, double* v, double* pos) {
	int i = blockIdx.x * blockIdx.x + threadIdx.x;
	double r_sqr;
	for (int j = 0; j < N; j++) {
		r_sqr = (pos[i] - pos[j]) * (pos[i] - pos[j])
			+ (pos[i + N] - pos[j + N]) * (pos[i + N] - pos[j + N])
			+ (pos[i + N * 2] - pos[j + N * 2]) * (pos[i + N * 2] - pos[j + N * 2]);

		a[i * N + j] = G * (m[j]) / (r_sqr)
			* (pos[i] - pos[j]) / (sqrt(r_sqr));
		a[i * N + j + N * N] = G * (m[j]) / (r_sqr)
			* (pos[i + N] - pos[j + N]) / (sqrt(r_sqr));
		a[i * N + j + N * N * 2] = G * (m[j]) / (r_sqr)
			* (pos[i + N * 2] - pos[j + N * 2]) / (sqrt(r_sqr));
	}

	for (int j = 0; j < N; j++) {
		if (i != j) {
			v[i] += DT * a[i * N + j];
			v[i + N] += DT * a[i * N + j + N * N];
			v[i + N * 2] += DT * a[i * N + j + N * N * 2];
		}
	}

	__syncthreads();

	pos[i] += DT * v[i];
	pos[i + N] += DT * v[i + N];
	pos[i + N * 2] += DT * v[i + N * 2];

}