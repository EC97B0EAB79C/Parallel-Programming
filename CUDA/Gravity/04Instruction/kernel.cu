#include "hip/hip_runtime.h"
﻿/*
CUDA code for making improvements by optimizing instructions

TODO:
	Compare initializing vs calculating: N2, NN, NN2
	Check N2, NN, NN2 variables
*/
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "../data_util/data_util.h"

#define N 10000
#define G 1
#define DT 1

__global__ void kernelAcceleration(double*, double*, double*, double*, double*);
__global__ void kernelVelocity(double*, double*, double*, double*, double*);
__global__ void kernelPosition(double*, double*, double*, double*, double*);
__global__ void kernelInit();

__device__ int N2;
__device__ int NN;
__device__ int NN2;

hipError_t runKernel(double* m, double* a, double* v, double* pos) {


	dim3 blockDim;
	dim3 gridDim;
	float milisecondsStoM = 0;
	float milisecondsStoE = 0;
	int temp;

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Err: hipSetDevice\n");
		goto Error;
	}

	size_t size;


	// device(CUDA) variable
	double* d_m;
	size = sizeof(double) * N;
	cudaStatus = hipMalloc(&d_m, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Err: hipMalloc\n");
		goto Error;
	}

	double* d_a;
	size = sizeof(double) * N * N * 3;
	cudaStatus = hipMalloc(&d_a, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Err: hipMalloc\n");
		goto Error;
	}

	double* d_v;
	size = sizeof(double) * N * 3;
	cudaStatus = hipMalloc(&d_v, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Err: hipMalloc\n");
		goto Error;
	}

	double* d_pos;
	cudaStatus = hipMalloc(&d_pos, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Err: hipMalloc\n");
		goto Error;
	}

	double* d_result;
	cudaStatus = hipMalloc(&d_result, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Err: hipMalloc\n");
		goto Error;
	}


	// performance metric
	hipEvent_t start, memcpy, end;
	hipEventCreate(&start);
	hipEventCreate(&memcpy);
	hipEventCreate(&end);
	hipEventRecord(start);	// record event

	// hipMemcpy
	size = sizeof(double) * N;
	cudaStatus = hipMemcpy(d_m, m, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Err: hipMemcpy\n");
		goto Error;
	}

	size = sizeof(double) * N * 3;
	cudaStatus = hipMemcpy(d_v, v, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Err: hipMemcpy\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(d_pos, pos, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Err: hipMemcpy\n");
		goto Error;
	}
	// record event
	hipEventRecord(memcpy);


	// launch kernel
	kernelInit<<<1, 1>>>();
	for (int i = 0; i < 10; i++) {
		blockDim = dim3(16, 16);
		gridDim = dim3((N + (blockDim.x - 1)) / blockDim.x, (N + (blockDim.y - 1)) / blockDim.y);
		kernelAcceleration <<<gridDim, blockDim>>> (d_m, d_a, d_v, d_pos, d_result);
		/*
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Err: %dth iter Kernel\n", i);
			goto Error;
		}
		*/

		blockDim = dim3(256);
		gridDim = dim3((N + (blockDim.x - 1)) / blockDim.x);
		kernelVelocity <<<gridDim, blockDim >>> (d_m, d_a, d_v, d_pos, d_result);
		/*
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Err: %dth iter Kernel\n", i);
			goto Error;
		}
		*/

		kernelPosition <<<gridDim, blockDim >>> (d_m, d_a, d_v, d_pos, d_result);
		/*
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Err: %dth iter Kernel\n", i);
			goto Error;
		}
		*/

		cudaStatus = hipMemcpy(d_pos, d_result, size, hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Err: %dth iter Kernel\n", i);
			goto Error;
		}
	}


	// hipMemcpy result from device
	hipDeviceSynchronize();
	cudaStatus = hipMemcpy(pos, d_pos, size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Err: hipMemcpy\n");
		goto Error;
	}
	hipEventRecord(end);	// record event
	hipDeviceSynchronize();

	// print performace metrics
	hipEventElapsedTime(&milisecondsStoM, start, memcpy);
	fprintf(stdout, "Time from start to memcpy:\n\t%f\n", milisecondsStoM);
	hipEventElapsedTime(&milisecondsStoE, start, end);
	fprintf(stdout, "Time from start to end:\n\t%f\n", milisecondsStoE);

Error:
	// free cuda memory
	hipFree(d_m);
	hipFree(d_a);
	hipFree(d_v);
	hipFree(d_pos);
	hipFree(d_result);

	hipDeviceReset();

	return cudaStatus;
}


int main() {
	size_t size;


	// host variable
	size = sizeof(double) * N;
	double* m = (double*)malloc(size);
	size = sizeof(double) * N * N * 3;
	double* a = (double*)malloc(size);
	size = sizeof(double) * N * 3;
	double* v = (double*)malloc(size);
	double* pos = (double*)malloc(size);


	// check variable
	if (
		m == NULL ||
		a == NULL ||
		v == NULL ||
		pos == NULL
		) {
		fprintf(stderr, "Err: Malloc Failed\n");
		return -1;
	}


	// set variable
	if (readDouble("../test/n10000/m.double", m) != N) {
		fprintf(stderr, "Err: Can not read m.double\n");
		return -1;
	}

	size = sizeof(double) * N * N * 3;
	memset(a, 0, size);

	size = sizeof(double) * N * 3;
	memset(v, 0, size);

	if (readDouble("../test/n10000/x.double", pos) != N) {
		fprintf(stderr, "Err: Can not read x.double\n");
		return -1;
	}
	if (readDouble("../test/n10000/y.double", pos + N) != N) {
		fprintf(stderr, "Err: Can not read y.double\n");
		return -1;
	}
	if (readDouble("../test/n10000/z.double", pos + N * 2) != N) {
		fprintf(stderr, "Err: Can not read z.double\n");
		return -1;
	}


	// run kernel
	hipError_t cudaStatus;
	if ((cudaStatus = runKernel(m, a, v, pos)) != hipSuccess) {
		fprintf(stderr, "Err: Kernal returned error code %d\n\t%s\n",
			cudaStatus,
			hipGetErrorString(cudaStatus));
	}

	//printf("%lf", pos[9999]);
	// write results
	char writeFileX[] = "./cuda_x.double";
	writeDouble(writeFileX, pos, N);
	char writeFileY[] = "./cuda_y.double";
	writeDouble(writeFileY, pos + N, N);
	char writeFileZ[] = "./cuda_z.double";
	writeDouble(writeFileZ, pos + N * 2, N);


	// free memory
	free(m);
	free(a);
	free(v);
	free(pos);

	return 0;
}

__global__ void kernelAcceleration(double* m, double* a, double* v, double* pos, double* result) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i >= N || j >= N) {
		return;
	}

	double r_sqr;
	double r3;
	
	r_sqr
		= (pos[i] - pos[j]) * (pos[i] - pos[j])
		+ (pos[i + N] - pos[j + N]) * (pos[i + N] - pos[j + N])
		+ (pos[i + N2] - pos[j + N2]) * (pos[i + N2] - pos[j + N2]);
	r3 = sqrt(r_sqr) * r_sqr;
	if(r3==0) return;
	a[i * N + j]
		= G * (m[j]) * (pos[j] - pos[i]) / r3;
	a[i * N + j + NN]
		= G * (m[j]) * (pos[j + N] - pos[i + N]) / r3;
	a[i * N + j + NN2]
		= G * (m[j]) * (pos[j + N2] - pos[i + N2]) / r3;
}

__global__ void kernelVelocity(double* m, double* a, double* v, double* pos, double* result) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= N) {
		return;
	}

	for (int j = 0; j < N; j++) {
		v[i] += DT * a[i * N + j];
		v[i + N] += DT * a[i * N + j + NN];
		v[i + N2] += DT * a[i * N + j + NN2];
	}
}

__global__ void kernelPosition(double* m, double* a, double* v, double* pos, double* result) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= N) {
		return;
	}

	result[i] = pos[i] + DT * v[i];
	result[i + N] = pos[i + N] + DT * v[i + N];
	result[i + N2] = pos[i + N2] + DT * v[i + N2];
}

__global__ void kernelInit(){
	N2 = N << 1;
	NN = N * N;
	NN2 = NN << 1;
}
